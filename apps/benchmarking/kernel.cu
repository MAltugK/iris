
#include <hip/hip_runtime.h>
extern "C" __global__ void saxpy(float* Z, float A, float* X, float* Y) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  Z[i] = A * X[i] + Y[i];
}

extern "C" __global__ void mem1(int l, float* m1) {
  for (int loop = 0; loop < l; loop++) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  m1[i] = i;
  }
}

extern "C" __global__ void mem2(int l, float* m1, float* m2) {
  for (int loop = 0; loop < l; loop++) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  m1[i] = i + m2[i];
  }
}

extern "C" __global__ void mem3(int l, float* m1, float* m2, float* m3) {
  for (int loop = 0; loop < l; loop++) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  m1[i] = i + m2[i] + m3[i];
  }
}

extern "C" __global__ void mem4(int l, float* m1, float* m2, float* m3, float* m4) {
  for (int loop = 0; loop < l; loop++) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  m1[i] = i + m2[i] + m3[i] + m4[i];
  }
}

extern "C" __global__ void mem5(int l, float* m1, float* m2, float* m3, float* m4, float* m5) {
  for (int loop = 0; loop < l; loop++) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  m1[i] = i + m2[i] + m3[i] + m4[i] + m5[i];
  }
}

extern "C" __global__ void mem6(int l, float* m1, float* m2, float* m3, float* m4, float* m5, float* m6) {
  for (int loop = 0; loop < l; loop++) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  m1[i] = i + m2[i] + m3[i] + m4[i] + m5[i] + m6[i];
  }
}

extern "C" __global__ void mem7(int l, float* m1, float* m2, float* m3, float* m4, float* m5, float* m6, float* m7) {
  for (int loop = 0; loop < l; loop++) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  m1[i] = i + m2[i] + m3[i] + m4[i] + m5[i] + m6[i] + m7[i];
  }
}

extern "C" __global__ void mem8(int l, float* m1, float* m2, float* m3, float* m4, float* m5, float* m6, float* m7, float* m8) {
  for (int loop = 0; loop < l; loop++) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  m1[i] = i + m2[i] + m3[i] + m4[i] + m5[i] + m6[i] + m7[i] + m8[i];
  }
}

extern "C" __global__ void mem9(int l, float* m1, float* m2, float* m3, float* m4, float* m5, float* m6, float* m7, float* m8, float* m9) {
  for (int loop = 0; loop < l; loop++) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  m1[i] = i + m2[i] + m3[i] + m4[i] + m5[i] + m6[i] + m7[i] + m8[i] + m9[i];
  }
}

extern "C" __global__ void mem10(int l, float* m1, float* m2, float* m3, float* m4, float* m5, float* m6, float* m7, float* m8, float* m9, float* m10) {
  for (int loop = 0; loop < l; loop++) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  m1[i] = i + m2[i] + m3[i] + m4[i] + m5[i] + m6[i] + m7[i] + m8[i] + m9[i] + m10[i];
  }
}

